#define _CRT_SECURE_NO_WARNINGS

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <omp.h>

#define FAILURE 0
#define SUCCESS !FAILURE

#define USER_NAME "aca15fm"		//replace with your user name

void print_help();
int process_command_line(int argc, char *argv[]);
int process_output_file(int tile_size);
int process_to_mosaic(int tile_size);
int process_ppm_file(FILE *file, int total_size);
int process_ppm_header(FILE *file);
int do_cpu(FILE *file, int tilse_size);
int do_cuda_processing(int height, int width, int tile_size);
void transform_1D_to_2D(unsigned char* out_array_r, unsigned char* out_array_g, unsigned char* out_array_b, int width, int height);

int IS_BINARY_MODE = 0;

int average_r = 0, average_g = 0, average_b = 0;
__device__ int gpu_average_r, gpu_average_g, gpu_average_b;

double start_timer, timer;

typedef enum MODE { CPU, OPENMP, CUDA, ALL } MODE;
MODE execution_mode;

unsigned int tile_size = 0;
char *file_name, *output_file;

typedef struct {
	unsigned char red, green, blue;
} PPMPixel;

typedef struct {
	unsigned int *red, *green, *blue;
} device_PPMPixel;

typedef struct {
	int height, width, maxval;
	PPMPixel *data;
	device_PPMPixel *device_data;
} PPMImage;

PPMImage *image;
PPMPixel **image_array; // Stores all the R,G,B values in an array the same size as the original image
PPMPixel **tile_array; // Stores the average RGB values of the Mosaic produced from the image
unsigned char *image_array_r, *image_array_g, *image_array_b;
unsigned char *out_array_r, *out_array_g, *out_array_b;

/* --------------------------------------------------
This functions takes in a ppm file and reads in the
header of the file. It stores the values of the header
in a PPMImage struct and sets the variable IS_BINARY_MODE
to 1 if the magic number is P6
-------------------------------------------------- */
int process_ppm_header(FILE *file) {
	char buff[16];

	//Open the file

	if (file == NULL) {
		printf("File is null");
		return FAILURE;
	}

	image = (PPMImage *)malloc(sizeof(PPMImage));
	if (!image) {
		free(image);
		printf("Error allocating memory for struct");
		return FAILURE;
	}

	// Scan magic number
	if (!fgets(buff, sizeof(buff), file)) {
		//if (fscanf(file, "%s", &image->magic_number) != 1) {
		printf("Magic number error");
		return FAILURE;
	}

	if (buff[0] == 'P' && buff[1] == '6')
		IS_BINARY_MODE = 1;

	// find comments and consume up to newline
	int comment = getc(file);
	while (comment == '#') {
		while (getc(file) != '\n');
		comment = getc(file); // consume the newline
	}

	printf(buff);
	ungetc(comment, file);
	//read size info

	if (fscanf(file, "%d %d", &image->width, &image->height) != 2) {
		return FAILURE;
	}

	if (fscanf(file, "%d", &image->maxval) != 1) {
		printf("Couldnt find colour maxval");
		return FAILURE;
	}

	while (getc(file) != '\n');

	return SUCCESS;
}

/* --------------------------------------------------
This functions takes in a file and the products of the
images width and height. It allocates memory for storing
the image as an array of PPMPixels and analyzes the file
to produce such array.
-------------------------------------------------- */
int process_ppm_file(FILE *file, int total_size) {
	// Check if one of the arguments passed in is null and return FAILURE
	if (file == NULL || total_size == NULL)
		return FAILURE;

	//printf("Processing ppm file \n");

	//int total_r = 0, total_g = 0, total_b = 0;
	int height = *(&image->height);
	int width = *(&image->width);

	// Allocate memory for 2D array of PPMPixel struct
	image_array = (PPMPixel **)malloc(width * height * sizeof(PPMPixel *));
	for (int i = 0; i < *(&image->width); i++) {
		image_array[i] = (PPMPixel *)malloc(width * sizeof(PPMPixel));
	}

	// Scan file and parse R,G,B values into PPMPixel struct
	for (int i = 0; i < width; i++) {
		for (int j = 0; j < height; j++) {
			// Allocate memory for a certain pixel and parse the R,G,B values
			PPMPixel *pixel = (PPMPixel *)malloc(sizeof(PPMPixel));
			if (pixel == NULL) {
				printf("pixel structure is null");
				return FAILURE;
			}
			// READ PIXEL VALUES, CHECK FOR BINARY MODE OR PLAIN TEXT MODE
			if (IS_BINARY_MODE == 1) {
				pixel->red = fgetc(file);
				pixel->green = fgetc(file);
				pixel->blue = fgetc(file);
			}
			else {
				fscanf(file, "%u %u %u\t", &pixel->red, &pixel->green, &pixel->blue);
			}
			// Store pointer to the pixel in the array
			image_array[i][j] = *(pixel);
		}
	}

	return SUCCESS;
}

int convert_struct_into_arrays() {
	int height = *(&image->height);
	int width = *(&image->width);

	image_array_r = (unsigned char *)malloc(height * width * sizeof(unsigned char));
	image_array_g = (unsigned char *)malloc(height * width * sizeof(unsigned char));
	image_array_b = (unsigned char *)malloc(height * width * sizeof(unsigned char));
	//image_array_1d = (unsigned char *)malloc(height * width * sizeof(PPMPixel));
	// Allocate memory for a temporary 2D array to average colour of pixels

	int count = 0;
	for (int i = 0; i < width; i++) {
		for (int j = 0; j < height; j++) {
			image_array_r[count] = image_array[i][j].red;
			image_array_g[count] = image_array[i][j].green;
			image_array_b[count] = image_array[i][j].blue;
			//printf("r %d, g %d, b %d \n", image_array[i][j].red, image_array[i][j].green, image_array[i][j].blue);
			//printf("r %d, g %d, b %d \n", image_array_r[count], image_array_g[count],image_array_b[count]);
			count++;
		}
	}
	return SUCCESS;
}

int get_average_color_values(FILE *file) {
	int total_r = 0, total_g = 0, total_b = 0;
	int height = *(&image->height);
	int width = *(&image->width);
	int total_size = width * height;
	for (int i = 0; i < width; i++) {
		for (int j = 0; j < height; j++) {
			// Add the R,G,B values to a counter
			total_r += image_array[i][j].red;
			total_g += image_array[i][j].green;
			total_b += image_array[i][j].blue;
		}
	}
	average_r = total_r / total_size;
	average_g = total_g / total_size;
	average_b = total_b / total_size;

	return SUCCESS;
}

/* --------------------------------------------------
This functions takes in the tile size and produces a
copy of the image_array of size (width/tile_size and
height/tile_size) with its pixels being a average for
the pixels in the original array
-------------------------------------------------- */
int process_to_mosaic(int tile_size) {
	const int height = *(&image->height) / tile_size;
	const int width = *(&image->width) / tile_size;
	//printf("Initial width %d height %d, mosaic width %d height %d \n", *(&image->width), *(&image->height), width, height);

	// Allocate memory for a temporary 2D array to average colour of pixels
	tile_array = (PPMPixel **)malloc(width * height * sizeof(PPMPixel *));
	for (int i = 0; i < *(&image->width); i++)
		tile_array[i] = (PPMPixel *)malloc(width * sizeof(PPMPixel));

	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			// Multiply i and j by the tile size to skip to the desired pixel
			int new_i = i*tile_size;
			int new_j = j*tile_size;
			int total_r = 0, total_g = 0, total_b = 0;
			for (int i2 = 0; i2 < tile_size; i2++) {
				for (int j2 = 0; j2 < tile_size; j2++) {
					total_r += image_array[new_i + i2][new_j + j2].red;
					total_g += image_array[new_i + i2][new_j + j2].green;
					total_b += image_array[new_i + i2][new_j + j2].blue;
					//printf("%d %d %d \n", image_array[new_i + i2][new_j + j2].red, image_array[new_i + i2][new_j + j2].green, image_array[new_i + i2][new_j + j2].blue);
				}
			}

			// Create a pixel struc and store it in the tile array
			PPMPixel *pixel = (PPMPixel *)malloc(sizeof(PPMPixel));
			pixel->red = total_r / (tile_size * tile_size);
			pixel->green = total_g / (tile_size * tile_size);
			pixel->blue = total_b / (tile_size * tile_size);
			tile_array[i][j] = *pixel;
			//printf("Average for pixels: %d %d %d \n", tile_array[i][j].red, tile_array[i][j].green, tile_array[i][j].blue);

			//printf("------- \n");
		}
	}
	return SUCCESS;
}

/* --------------------------------------------------
This functions takes in the tile size value and
loops over the tile_array to get the colours for
each pixel. Then it creates a ppm file with the dimensions
of the original image.
-------------------------------------------------- */
int process_output_file(int tile_size) {
	FILE *out_file;
	out_file = fopen(output_file, "wb");

	/*-----------------------------
	Output the file header
	-----------------------------*/
	if (IS_BINARY_MODE == 1) {
		fprintf(out_file, "%s\n", "P6");
	}
	else {
		fprintf(out_file, "%s\n", "P3");
	}
	fprintf(out_file, "%d\n", *(&image->width));
	fprintf(out_file, "%d\n", *(&image->height));
	fprintf(out_file, "%d\n", *(&image->maxval));

	// Calculate te size of the mosaic image
	const int height = *(&image->height) / tile_size;
	const int width = *(&image->width) / tile_size;
	// printf("Initial width %d height %d, mosaic width %d height %d \n", *(&image->width), *(&image->height), width, height);

	if (tile_array == NULL) {
		printf("tile_array is NULL");
		return FAILURE;
	}

	for (int i = 0; i < height; i++) {
		for (int i2 = 0; i2 < tile_size; i2++) {
			for (int j = 0; j < width; j++) {
				for (int j2 = 0; j2 < tile_size; j2++) {
					// Multiply i and j by the tile size to skip to the desired pixel
					if (&tile_array[i][j] == NULL)
						return FAILURE;

					if (IS_BINARY_MODE == 1) {
						fwrite(&tile_array[i][j], sizeof(PPMPixel), 1, out_file);
					}
					else {
						fprintf(out_file, "%d %d %d\t", tile_array[i][j].red, tile_array[i][j].green, tile_array[i][j].blue);
					}
					//printf("%d %d %d ", tile_array[i][j].red, tile_array[i][j].green, tile_array[i][j].blue);
				}
			}
			if (IS_BINARY_MODE == 0) {
				fprintf(out_file, "\n");
			}
			//printf("\n");
		}
	}
	fclose(out_file);

	return SUCCESS;
}

int cuda_process_output_file(int tile_size) {
	FILE *out_file;
	out_file = fopen(output_file, "wb");

	/*-----------------------------
	Output the file header
	-----------------------------*/
	if (IS_BINARY_MODE == 1) {
		fprintf(out_file, "%s\n", "P6");
	}
	else {
		fprintf(out_file, "%s\n", "P3");
	}
	fprintf(out_file, "%d\n", *(&image->width));
	fprintf(out_file, "%d\n", *(&image->height));
	fprintf(out_file, "%d\n", *(&image->maxval));

	// Calculate te size of the mosaic image
	const int height = *(&image->height);
	const int width = *(&image->width);
	// printf("Initial width %d height %d, mosaic width %d height %d \n", *(&image->width), *(&image->height), width, height);

	if (tile_array == NULL) {
		printf("tile_array is NULL");
		return FAILURE;
	}

	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			// Multiply i and j by the tile size to skip to the desired pixel
			if (&tile_array[i][j] == NULL)
				return FAILURE;

			if (IS_BINARY_MODE == 1) {
				fwrite(&tile_array[i][j], sizeof(PPMPixel), 1, out_file);
			}
			else {
				fprintf(out_file, "%d %d %d\t", tile_array[i][j].red, tile_array[i][j].green, tile_array[i][j].blue);
			}
			//printf("%d,%d:%d %d %d ", i, j, tile_array[i][j].red, tile_array[i][j].green, tile_array[i][j].blue);
				
		}
		if (IS_BINARY_MODE == 0) {
			fprintf(out_file, "\n");
		}
		//printf("\n");
	}
	fclose(out_file);

	return SUCCESS;
}

/* --------------------------------------------------
This functions takes in a file and the products of the
images width and height. It allocates memory for storing
the image as an array of PPMPixels and analyzes the file
to produce such array.
-------------------------------------------------- */
int openmp_process_ppm_file(FILE* file, int total_size) {
	// Check if one of the arguments passed in is null and return FAILURE
	if (file == 0 || total_size == NULL)
		return FAILURE;

	//printf("Processing ppm file \n");

	//int total_r = 0, total_g = 0, total_b = 0;
	int height = *(&image->height);
	int width = *(&image->width);

	// Allocate memory for 2D array of PPMPixel struct
	image_array = (PPMPixel **)malloc(width * height * sizeof(PPMPixel *));
	int i, j;
	for (i = 0; i < *(&image->width); i++)
		image_array[i] = (PPMPixel *)malloc(width * sizeof(PPMPixel));

	// Scan file and parse R,G,B values into PPMPixel struct

	for (i = 0; i < width; i++) {
		for (j = 0; j < height; j++) {
			// Allocate memory for a certain pixel and parse the R,G,B values
			PPMPixel *pixel = (PPMPixel *)malloc(sizeof(PPMPixel));
			if (pixel == NULL) {
				printf("pixel structure is null");
				return FAILURE;
			}
			// READ PIXEL VALUES, CHECK FOR BINARY MODE OR PLAIN TEXT MODE
			if (IS_BINARY_MODE == 1) {
				pixel->red = fgetc(file);
				pixel->green = fgetc(file);
				pixel->blue = fgetc(file);
			}
			else {
				fscanf(file, "%u %u %u\t", &pixel->red, &pixel->green, &pixel->blue);
			}
			// Store pointer to the pixel in the array
			image_array[i][j] = *(pixel);
		}
	}

	return SUCCESS;
}

int openmp_get_average_color_values(FILE *file) {
	int total_r = 0, total_g = 0, total_b = 0;
	int height = *(&image->height);
	int width = *(&image->width);
	int total_size = width * height;
	int i, j;

#pragma omp parallel for private(i, j) shared(image_array, width, height) reduction(+: total_r, total_g, total_b) schedule(static)
	// #pragma omp for nowait
	for (i = 0; i < width; i++) {
		for (j = 0; j < height; j++) {
			// Add the R,G,B values to a counter
			total_r += image_array[i][j].red;
			total_g += image_array[i][j].green;
			total_b += image_array[i][j].blue;
		}
	}
	average_r = total_r / total_size;
	average_g = total_g / total_size;
	average_b = total_b / total_size;
	// printf("Average R,G,B values: %d %d %d \n", total_r, total_g, total_b);

	return SUCCESS;
}

/* --------------------------------------------------
This functions takes in the tile size and produces a
copy of the image_array of size (width/tile_size and
height/tile_size) with its pixels being a average for
the pixels in the original array
-------------------------------------------------- */
int openmp_process_to_mosaic(int tile_size) {
	const int height = (int)(*(&image->height) / tile_size);
	const int width = (int)(*(&image->width) / tile_size);
	//printf("Initial width %d height %d, mosaic width %d height %d \n", *(&image->width), *(&image->height), width, height);

	// Allocate memory for a temporary 2D array to average colour of pixels
	tile_array = (PPMPixel **)malloc(width * height * sizeof(PPMPixel *));
	for (int i = 0; i < *(&image->width); i++)
		tile_array[i] = (PPMPixel *)malloc(width * sizeof(PPMPixel));

	int i, j, new_i, new_j, i2, j2;
	int total_r = 0, total_g = 0, total_b = 0;
#pragma omp parallel private(i, j, new_i, new_j, i2, j2) shared(image_array, tile_size)
	{
		//#pragma omp for reduction(+: total_r, total_g, total_b) schedule(static)
		for (i = 0; i < height; i++) {
			for (j = 0; j < width; j++) {
				// Multiply i and j by the tile size to skip to the desired pixel
				new_i = i*tile_size;
				new_j = j*tile_size;
				total_r = 0, total_g = 0, total_b = 0;
				for (i2 = 0; i2 < tile_size; i2++) {
					for (j2 = 0; j2 < tile_size; j2++) {
						if ((new_i + i2) > image->height || new_j + j2 > image->width) {

						}
						else {
							total_r += image_array[new_i + i2][new_j + j2].red;
							total_g += image_array[new_i + i2][new_j + j2].green;
							total_b += image_array[new_i + i2][new_j + j2].blue;
						}
						//printf("%d %d %d \n", image_array[new_i + i2][new_j + j2].red, image_array[new_i + i2][new_j + j2].green, image_array[new_i + i2][new_j + j2].blue);
					}
				}

				// Create a pixel struc and store it in the tile array
				tile_array[i][j].red = total_r / (tile_size * tile_size);
				tile_array[i][j].green = total_g / (tile_size * tile_size);
				tile_array[i][j].blue = total_b / (tile_size * tile_size);
				//printf("Average for pixels: %d %d %d \n", tile_array[i][j].red, tile_array[i][j].green, tile_array[i][j].blue);

				//printf("------- \n");
			}
		}
	}
	return SUCCESS;
}

/* --------------------------------------------------
//This functions takes in the tile size value and
//loops over the tile_array to get the colours for
//each pixel. Then it creates a ppm file with the dimensions
//of the original image.
-------------------------------------------------- */
int openmp_process_output_file(int tile_size) {
	FILE *out_file;
	out_file = fopen(output_file, "wb");

	/*-----------------------------
	Output the file header
	-----------------------------*/
	if (IS_BINARY_MODE == 1) {
		fprintf(out_file, "%s\n", "P6");
	}
	else {
		fprintf(out_file, "%s\n", "P3");
	}
	fprintf(out_file, "%d\n", *(&image->width));
	fprintf(out_file, "%d\n", *(&image->height));
	fprintf(out_file, "%d\n", *(&image->maxval));

	// Calculate te size of the mosaic image
	const int height = *(&image->height) / tile_size;
	const int width = *(&image->width) / tile_size;
	// printf("Initial width %d height %d, mosaic width %d height %d \n", *(&image->width), *(&image->height), width, height);

	if (tile_array == NULL) {
		printf("tile_array is NULL");
		return FAILURE;
	}

	int i, i2, j, j2;
	for (i = 0; i < height; i++) {
		for (i2 = 0; i2 < tile_size; i2++) {
			for (j = 0; j < width; j++) {
				for (j2 = 0; j2 < tile_size; j2++) {
					// Multiply i and j by the tile size to skip to the desired pixel
					if (&tile_array[i][j] == NULL)
						return FAILURE;

					if (IS_BINARY_MODE == 1) {
						fwrite(&tile_array[i][j], sizeof(PPMPixel), 1, out_file);
					}
					else {
						fprintf(out_file, "%d %d %d\t", tile_array[i][j].red, tile_array[i][j].green, tile_array[i][j].blue);
					}
					//printf("%d %d %d ", tile_array[i][j].red, tile_array[i][j].green, tile_array[i][j].blue);
				}
			}
			if (IS_BINARY_MODE == 0) {
				fprintf(out_file, "\n");
			}
			//printf("\n");
		}
	}
	fclose(out_file);

	return SUCCESS;
}

void print_help() {
	printf("mosaic_%s C M -i input_file -o output_file [options]\n", USER_NAME);

	printf("where:\n");
	printf("\tC              Is the mosaic cell size which should be any positive\n"
		"\t               power of 2 number \n");
	printf("\tM              Is the mode with a value of either CPU, OPENMP, CUDA or\n"
		"\t               ALL. The mode specifies which version of the simulation\n"
		"\t               code should execute. ALL should execute each mode in\n"
		"\t               turn.\n");
	printf("\t-i input_file  Specifies an input image file\n");
	printf("\t-o output_file Specifies an output image file which will be used\n"
		"\t               to write the mosaic image\n");
	printf("[options]:\n");
	printf("\t-f ppm_format  PPM image output format either PPM_BINARY (default) or \n"
		"\t               PPM_PLAIN_TEXT\n ");

	getchar();
}

int process_command_line(int argc, char *argv[]) {
	if (argc < 7) {
		fprintf(stderr, "Error: Missing program arguments. Correct usage is...\n");
		print_help();
		return FAILURE;
	}

	//first argument is always the executable name

	//read in the non optional command line arguments
	tile_size = (unsigned int)atoi(argv[1]);
	int temp_c = tile_size;
	while ((temp_c % 2) == 0 && temp_c > 1)
		temp_c = temp_c / 2;
	if (temp_c != 1) {
		printf("C has to be equal to 2^n where N is a positive number");
		return(FAILURE);
	}

	//TODO: read in the mode
	if (strcmp(argv[2], "CPU") == 0) {
		execution_mode = CPU;
	}
	if (strcmp(argv[2], "OPENMP") == 0) {
		execution_mode = OPENMP;
	}
	if (strcmp(argv[2], "CUDA") == 0) {
		execution_mode = CUDA;
	}
	if (strcmp(argv[2], "ALL") == 0) {
		execution_mode = ALL;
	}

	//TODO: read in the input image name
	file_name = argv[4];
	//printf(argv[4]);

	//TODO: read in the output image name
	output_file = argv[6];
	//printf(argv[6]);

	return SUCCESS;
}

void transform_1D_to_2D(unsigned char* out_array_r, unsigned char* out_array_g, unsigned char* out_array_b, int width, int height) {
	int theCount = 0;

	tile_array = (PPMPixel **)malloc((*(&image->width)) * (*(&image->height)) * sizeof(PPMPixel *));
	for (int i = 0; i < (*(&image->width)); i++)
		tile_array[i] = (PPMPixel *)malloc((*(&image->width)) * sizeof(PPMPixel));

	// transfer 1D array to 2D
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			//printf("i %d, j %d, r %d, g %d, b %d \n", i, j, out_array_r[theCount], out_array_g[theCount], out_array_b[theCount]);
			tile_array[i][j].red = out_array_r[theCount];
			tile_array[i][j].green = out_array_g[theCount];
			tile_array[i][j].blue = out_array_b[theCount];
			theCount += 1;
		}
	}
}

__device__ unsigned long long gpu_total_r = 0, gpu_total_g = 0, gpu_total_b = 0;

__global__ void get_image_averages(uchar3* gpu_image, int width, int height, int c) {
	// each thread loads one element from global to shared mem
	//unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

	atomicAdd(&gpu_total_r, (unsigned long long) gpu_image[i].x);
	atomicAdd(&gpu_total_g, (unsigned long long) gpu_image[i].y);
	atomicAdd(&gpu_total_b, (unsigned long long) gpu_image[i].z);
}



__global__ void cuda_image_pixelize(uchar3* gpu_image, int width, int height, int tilesize) {
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int output_offset = x + y * blockDim.x * gridDim.x;
	float totalsize = (float)(tilesize*tilesize);
	if ((output_offset == 0 || output_offset % tilesize == 0) && (y == 0 || y % tilesize == 0)) {
		int avg_r = 0, avg_g = 0, avg_b = 0;
		for (int i = 0; i < tilesize; i++) {
			for (int j = 0; j < tilesize; j++) {
				int index = output_offset + i + (j*height);
				uchar3 pixel = gpu_image[index];
				avg_r += pixel.x;
				avg_g += pixel.y;
				avg_b += pixel.z;
			}
		}
		__syncthreads();
		for (int i = 0; i < tilesize; i++) {
			for (int j = 0; j < tilesize; j++) {
				int out_index = output_offset + i + (j * height);
				//printf("out: %d \n",out_index);
				gpu_image[out_index].x = (unsigned char)(avg_r / totalsize);
				gpu_image[out_index].y = (unsigned char)(avg_g / totalsize);
				gpu_image[out_index].z = (unsigned char)(avg_b / totalsize);

			}
		}
	}
}

int do_cuda_processing(int height, int width, int tile_size) {
	hipEvent_t start, stop;
	float mseconds;
	uchar3 *cpu_pixel;
	uchar3 *gpu_pixel;
	//cuda layout and execution
	dim3 blocksPerGrid2(width / 16, height / 16);
	dim3 threadsPerBlock2(16, 16);
	unsigned long long total_r, total_b, total_g;

	// Allocate memory for the arrays of R, G, B values that are going to be produced
	out_array_r = (unsigned char *)malloc((width)*(height) * sizeof(unsigned char));
	out_array_g = (unsigned char *)malloc((width)*(height) * sizeof(unsigned char));
	out_array_b = (unsigned char *)malloc((width)*(height) * sizeof(unsigned char));

	// Optimization: User uchar3 to keep rgb values coalesced in memory. 
	// We have to copy them from the three arrays into one
	cpu_pixel = (uchar3*)malloc(sizeof(uchar3)*(width)*(height));

	// Copy the r,g,b arrays into the uchar3
	for (int i = 0; i < width*height; i++) {
		cpu_pixel[i].x = image_array_r[i];
		cpu_pixel[i].y = image_array_g[i];
		cpu_pixel[i].z = image_array_b[i];
	}

	printf("width %d, height %d \n", width, height);
	// create timers
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// starting timing here
	hipEventRecord(start, 0);

	// Allocate memory in GPU
	hipMalloc((void**)&gpu_pixel, sizeof(uchar3)*(width)*(height));

	// Copy data into GPU memory
	hipMemcpy(gpu_pixel, cpu_pixel, sizeof(uchar3)*(width)*(height), hipMemcpyHostToDevice);

	printf("getting image average...\n");
	get_image_averages <<<blocksPerGrid2, threadsPerBlock2 >>>(gpu_pixel, width, height, tile_size);

	printf("pixelating image...\n");
	cuda_image_pixelize <<<blocksPerGrid2, threadsPerBlock2 >>>(gpu_pixel, width, height, tile_size);
	hipDeviceSynchronize();

	printf("done pixelating...\n");

	//cudaMemcpyToSymbol(&gpu_total_r, &total_r, sizeof(int));

	hipMemcpyFromSymbol(&total_r, HIP_SYMBOL(gpu_total_r), sizeof(unsigned long long));
	hipMemcpyFromSymbol(&total_g, HIP_SYMBOL(gpu_total_g), sizeof(unsigned long long));
	hipMemcpyFromSymbol(&total_b, HIP_SYMBOL(gpu_total_b), sizeof(unsigned long long));
	printf("done memcopy1\n");

	// Copy data back from gpu
	hipMemcpy(cpu_pixel, gpu_pixel, sizeof(uchar3)*(width)*(height), hipMemcpyDeviceToHost);

	// end timing here
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&mseconds, start, stop);

	printf("CUDA mode execution time took %d s and %d ms\n", (int)mseconds / 1000, (int)mseconds % 1000);

	printf("cpu_average_r total is %d \n", total_r / (width*height));
	printf("cpu_average_g total is %d \n", total_g / (width*height));
	printf("cpu_average_b total is %d \n", total_b / (width*height));

	// Improvement
	// Copy data back to the initial format so that the output function still works
	for (int i = 0; i < width*height; i++) {
		out_array_r[i] = cpu_pixel[i].x;
		out_array_g[i] = cpu_pixel[i].y;
		out_array_b[i] = cpu_pixel[i].z;
	}

	// Free GPU memory
	hipFree(gpu_pixel);
	free(cpu_pixel);

	// cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipDeviceReset();
	return SUCCESS;
}

int do_cpu(FILE *file, int tilse_size) {
	//TODO: starting timing here
	clock_t begin, end;
	float mseconds;
	begin = clock();

	if (get_average_color_values(file) == FAILURE) {
		printf("There was a problem averaging the colours");
	}

	if (process_to_mosaic(tile_size) == FAILURE) {
		printf("There was a problem processing the output file");
	}

	// Output the average colour value for the image
	printf("CPU Average image colour red = %d, green = %d, blue = %d \n", average_r, average_g, average_b);

	//TODO: end timing here

	end = clock();
	mseconds = (end - begin) * 1000 / (float)CLOCKS_PER_SEC;
	printf("CPU mode execution time took %d s and %d ms\n", (int)mseconds / 1000, (int)mseconds % 1000);
	// starting timing here

	if (process_output_file(tile_size) == FAILURE) {
		printf("There was a problem processing the output file");
	}

	return SUCCESS;
}

int main(int argc, char *argv[]) {

	if (process_command_line(argc, argv) == FAILURE)
		return 1;

	FILE *file;
	file = fopen(file_name, "rb");

	if (process_ppm_header(file) == FAILURE) {
		printf("There was a problem processing the file header");
		exit(FAILURE);
	}

	int total_size = *(&image->width) * *(&image->height);
	if (process_ppm_file(file, total_size) != 1) {
		printf("There was a problem reading the pixels in the ppm file");
		exit(FAILURE);
	}

	if (tile_size > (unsigned int) *(&image->width) || tile_size >(unsigned int) *(&image->height)) {
		printf("You cant enter a mosaic size bigger than the actual image");
		exit(FAILURE);
	}

	//TODO: execute the mosaic filter based on the mode
	switch (execution_mode) {
		case (CPU): {
			do_cpu(file, tile_size);
			//TODO: starting timing here
			/*start_timer = omp_get_wtime();
			if (get_average_color_values(file) == FAILURE) {
				printf("There was a problem averaging the colours");
			}

			if (process_to_mosaic(tile_size) == FAILURE) {
				printf("There was a problem processing the output file");
			}

			// Output the average colour value for the image
			printf("CPU Average image colour red = %d, green = %d, blue = %d \n", average_r, average_g, average_b);

			//TODO: end timing here
			timer = omp_get_wtime() - start_timer;
			int seconds = (int)timer;
			double milisecs = (timer - seconds) * 1000;
			printf("CPU mode execution time took %d s and %f ms\n", seconds, milisecs);

			if (process_output_file(tile_size) == FAILURE) {
				printf("There was a problem processing the output file");
			}*/
			break;
		}
		case (OPENMP): {
			//TODO: starting timing here
			start_timer = omp_get_wtime();

			if (openmp_get_average_color_values(file) == FAILURE) {

			}
			if (openmp_process_to_mosaic(tile_size) == FAILURE) {
				printf("There was a problem processing the output file");
			}

			// Output the average colour value for the image
			printf("OPENMP Average image colour red = %d, green = %d, blue = %d \n", average_r, average_g, average_b);

			//TODO: end timing here
			timer = omp_get_wtime() - start_timer;
			int seconds = (int)timer;
			double milisecs = (timer - seconds) * 1000;
			printf("OPENMP mode execution time took %d s and %fms\n", seconds, milisecs);
			if (openmp_process_output_file(tile_size) == FAILURE) {
				printf("There was a problem processing the output file");
			}
			break;
		}
		case (CUDA): {
			printf("------------------------------- \n");
			printf("      Launching CUDA Mode \n");
			printf("------------------------------- \n");
			convert_struct_into_arrays();

			do_cuda_processing(*(&image->width), *(&image->height), tile_size);

			printf("transforming 1d to 2d \n");
			transform_1D_to_2D(out_array_r, out_array_g, out_array_b, *(&image->width), *(&image->height));

			printf("processing output file \n");
			if (cuda_process_output_file(tile_size) == FAILURE) {
				printf("There was a problem processing the output file");
			}
			break;
		}
		case (ALL): {
			//TODO: starting timing here
			printf("------------------------------- \n");
			printf("      Launching CPU Mode \n");
			printf("------------------------------- \n");
			start_timer = omp_get_wtime();
			if (get_average_color_values(file) == FAILURE) {
				printf("There was a problem averaging the colours");
			}

			if (process_to_mosaic(tile_size) == FAILURE) {
				printf("There was a problem processing the output file");
			}

			// Output the average colour value for the image
			printf("CPU Average image colour red = %d, green = %d, blue = %d \n", average_r, average_g, average_b);

			//TODO: end timing here
			timer = omp_get_wtime() - start_timer;
			int seconds = (int)timer;
			double milisecs = (timer - seconds) * 1000;
			printf("CPU mode execution time took %d s and %f ms\n", seconds, milisecs);

			if (process_output_file(tile_size) == FAILURE) {
				printf("There was a problem processing the output file");
			}

			printf("------------------------------- \n");
			printf("    Launching OPENMP Mode \n");
			printf("------------------------------- \n");
			//TODO: starting timing here
			start_timer = omp_get_wtime();

			if (openmp_get_average_color_values(file) == FAILURE) {

			}
			if (openmp_process_to_mosaic(tile_size) == FAILURE) {
				printf("There was a problem processing the output file");
			}

			// Output the average colour value for the image
			printf("OPENMP Average image colour red = %d, green = %d, blue = %d \n", average_r, average_g, average_b);

			//TODO: end timing here
			timer = omp_get_wtime() - start_timer;
			printf("OPENMP mode execution time took %d s and %fms\n", seconds, milisecs);
			if (openmp_process_output_file(tile_size) == FAILURE) {
				printf("There was a problem processing the output file");
			}

			printf("------------------------------- \n");
			printf("      Launching CUDA Mode \n");
			printf("------------------------------- \n");
			convert_struct_into_arrays();

			do_cuda_processing(*(&image->width), *(&image->height), tile_size);

			printf("transforming 1d to 2d \n");
			transform_1D_to_2D(out_array_r, out_array_g, out_array_b, *(&image->width), *(&image->height));

			printf("processing output file \n");
			if (cuda_process_output_file(tile_size) == FAILURE) {
				printf("There was a problem processing the output file");
			}
			break;
		}
	}

	free(image_array);

	free(tile_array);
	getchar();

	//save the output image file (from last executed mode)
	return 0;
}